#include "hip/hip_runtime.h"
#define NR_BITS 8
#define NR_CHANNELS 480
#define NR_POLARIZATIONS 2
#define NR_SAMPLES_PER_CHANNEL 3072
#define NR_RECEIVERS 576
#define NR_BASELINES ((NR_RECEIVERS) * ((NR_RECEIVERS) + 1) / 2)
#define NR_RECEIVERS_PER_BLOCK 64
#define NR_TIMES_PER_BLOCK (128 / (NR_BITS))


#include "test/Common/ComplexInt4.h"
#include "libtcc/Correlator.h"

#include <complex>
#include <iostream>

#include <hip/hip_runtime.h>
#if NR_BITS == 16
#include <hip/hip_fp16.h>
#endif


inline void checkCudaCall(hipError_t error)
{
  if (error != hipSuccess) {
    std::cerr << "error " << error << std::endl;
    exit(1);
  }
}


#if NR_BITS == 4
typedef complex_int4_t	      Sample;
typedef std::complex<int32_t> Visibility;
#elif NR_BITS == 8
typedef std::complex<int8_t>  Sample;
typedef std::complex<int32_t> Visibility;
#elif NR_BITS == 16
typedef std::complex<__half>  Sample;
typedef std::complex<float>   Visibility;
#endif

typedef Sample Samples[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK][NR_RECEIVERS][NR_POLARIZATIONS][NR_TIMES_PER_BLOCK];
typedef Visibility Visibilities[NR_CHANNELS][NR_BASELINES][NR_POLARIZATIONS][NR_POLARIZATIONS];


int main()
{
  try {
    checkCudaCall(hipSetDevice(0)); // combine the CUDA runtime API and CUDA driver API
    checkCudaCall(hipFree(0));

    tcc::Correlator correlator(NR_BITS, NR_RECEIVERS, NR_CHANNELS, NR_SAMPLES_PER_CHANNEL, NR_POLARIZATIONS, NR_RECEIVERS_PER_BLOCK);

    hipStream_t stream;
    Samples *samples;
    Visibilities *visibilities;

    checkCudaCall(hipStreamCreate(&stream));
    checkCudaCall(hipMallocManaged(&samples, sizeof(Samples)));
    checkCudaCall(hipMallocManaged(&visibilities, sizeof(Visibilities)));

    (*samples)[NR_CHANNELS / 3][NR_SAMPLES_PER_CHANNEL / 5 / NR_TIMES_PER_BLOCK][174][0][NR_SAMPLES_PER_CHANNEL / 5 % NR_TIMES_PER_BLOCK] = Sample(2, 3);
    (*samples)[NR_CHANNELS / 3][NR_SAMPLES_PER_CHANNEL / 5 / NR_TIMES_PER_BLOCK][418][0][NR_SAMPLES_PER_CHANNEL / 5 % NR_TIMES_PER_BLOCK] = Sample(4, 5);

    correlator.launchAsync((hipStream_t) stream, (hipDeviceptr_t) visibilities, (hipDeviceptr_t) samples);
    checkCudaCall(hipDeviceSynchronize());

    std::cout << ((*visibilities)[160][87745][0][0] == Visibility(23, -2) ? "success" : "failed") << std::endl;

    checkCudaCall(hipFree(visibilities));
    checkCudaCall(hipFree(samples));
    checkCudaCall(hipStreamDestroy(stream));
  } catch (std::exception &error) {
    std::cerr << error.what() << std::endl;
  }
}
